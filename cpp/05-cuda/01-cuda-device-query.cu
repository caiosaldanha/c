#include "hip/hip_runtime.h"
// This program queries and displays information about CUDA-enabled GPUs on the system.
//
// ================================= IMPORTANT ==================================
// TO COMPILE AND RUN THIS FILE:
// 1. CUDA Toolkit: You must have the NVIDIA CUDA Toolkit installed.
//    Download from https://developer.nvidia.com/cuda-downloads
// 2. NVIDIA GPU: You need an NVIDIA CUDA-enabled GPU.
// 3. NVIDIA Driver: Ensure you have a compatible NVIDIA driver installed.
// 4. Compiler: Use the NVIDIA CUDA Compiler (nvcc).
//
// COMPILE COMMAND (example):
//   nvcc 01-cuda-device-query.cu -o 01-cuda-device-query
//
// RUN COMMAND (example):
//   ./01-cuda-device-query
// ==============================================================================

#include <hip/hip_runtime.h> // CUDA Runtime API
#include <iostream>       // For std::cout, std::cerr
#include <cstdio>         // For printf (can be more convenient for formatting)

// --- CUDA Error Checking Macro ---
// This simple macro wraps CUDA API calls and checks for errors.
// More robust error handling might involve a function.
#define CUDA_CHECK(err) __cuda_check_errors((err), __FILE__, __LINE__)

inline void __cuda_check_errors(hipError_t err, const char *file, int line) {
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA error: %s (%d) in %s at line %d\n",
                hipGetErrorString(err), err, file, line);
        // It's common to exit or throw an exception upon CUDA error.
        // For this example, we'll just print and let it continue if possible,
        // or exit if critical (like no devices).
    }
}


int main() {
    std::cout << "--- CUDA Device Query ---" << std::endl;

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed! CUDA driver and runtime version may be mismatched.\n");
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error_id));
        if (error_id == hipErrorInsufficientDriver) {
             fprintf(stderr, "Ensure you have a recent NVIDIA driver installed.\n");
        }
        return 1; // Critical error, cannot proceed
    }

    CUDA_CHECK(error_id); // Check for other potential errors from hipGetDeviceCount

    if (deviceCount == 0) {
        std::cout << "No CUDA-enabled devices found on this system." << std::endl;
        return 0; // Not an error, but nothing to query
    }

    std::cout << "Found " << deviceCount << " CUDA-enabled device(s)." << std::endl;
    std::cout << "------------------------------------------------------" << std::endl;

    // Loop through each device
    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp; // Structure to hold device properties
        CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev)); // Get properties for device 'dev'

        printf("Device %d: \"%s\"\n", dev, deviceProp.name);

        // Compute Capability (major.minor)
        // This defines the features supported by the GPU hardware.
        // Higher numbers generally mean more features and better performance.
        printf("  Compute Capability:        %d.%d\n", deviceProp.major, deviceProp.minor);

        // Total Global Memory
        // This is the total amount of memory available on the GPU's dedicated RAM (e.g., GDDR6).
        printf("  Total Global Memory:       %.0f MiB (%llu bytes)\n",
               static_cast<float>(deviceProp.totalGlobalMem) / (1024.0f * 1024.0f),
               (unsigned long long)deviceProp.totalGlobalMem);

        // Shared Memory Per Block
        // Fast on-chip memory shared by threads within a block. Crucial for performance.
        printf("  Shared Memory Per Block:   %zu bytes\n", deviceProp.sharedMemPerBlock);

        // Registers Per Block
        printf("  Registers Per Block:       %d\n", deviceProp.regsPerBlock);

        // Warp Size
        // Threads on a CUDA GPU execute in groups called warps. This is typically 32.
        printf("  Warp Size:                 %d threads\n", deviceProp.warpSize);

        // Max Threads Per Block
        // The maximum number of threads that can be launched in a single execution block.
        printf("  Max Threads Per Block:     %d\n", deviceProp.maxThreadsPerBlock);

        // Max Threads Per Multiprocessor
        printf("  Max Threads Per SM:        %d\n", deviceProp.maxThreadsPerMultiProcessor);

        // Max Dimensions of a Block (x, y, z)
        // Threads in a block can be organized in 1D, 2D, or 3D. These are the max sizes.
        printf("  Max Block Dimensions:      %d x %d x %d\n",
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);

        // Max Dimensions of a Grid (x, y, z)
        // Blocks are organized into a grid. These are the max dimensions of the grid.
        printf("  Max Grid Dimensions:       %d x %d x %d\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

        // Clock Rate
        printf("  GPU Clock Rate:            %.0f MHz (%0.2f GHz)\n",
                deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

        // Memory Clock Rate
        printf("  Memory Clock Rate:         %.0f MHz\n", deviceProp.memoryClockRate * 1e-3f);

        // Memory Bus Width
        printf("  Memory Bus Width:          %d-bit\n", deviceProp.memoryBusWidth);

        // L2 Cache Size
        if (deviceProp.l2CacheSize > 0) {
            printf("  L2 Cache Size:             %d bytes\n", deviceProp.l2CacheSize);
        }

        // Number of Multiprocessors (SMs)
        printf("  Number of Multiprocessors: %d\n", deviceProp.multiProcessorCount);

        // CUDA Cores (approximate, depends on architecture)
        // This calculation is a common heuristic but not officially defined by NVIDIA as "CUDA Cores" in deviceProp.
        // It's derived from SM count and cores per SM for different architectures.
        // For Kepler: 192 cores/SM. Maxwell: 128. Pascal: 64 or 128. Volta/Turing/Ampere: 64 FP32 cores/SM.
        int cudaCores = 0;
        if (deviceProp.major == 2) { // Fermi
            if (deviceProp.minor == 1) cudaCores = deviceProp.multiProcessorCount * 48;
            else cudaCores = deviceProp.multiProcessorCount * 32;
        } else if (deviceProp.major == 3) { // Kepler
            cudaCores = deviceProp.multiProcessorCount * 192;
        } else if (deviceProp.major == 5) { // Maxwell
            cudaCores = deviceProp.multiProcessorCount * 128;
        } else if (deviceProp.major == 6) { // Pascal
            if (deviceProp.minor == 0 || deviceProp.minor == 1) cudaCores = deviceProp.multiProcessorCount * 64; // GP100, GP104 might have 64 or 128 (e.g. P100 vs 1080)
            else cudaCores = deviceProp.multiProcessorCount * 128; // GP102, GP104, etc often 128
        } else if (deviceProp.major == 7) { // Volta / Turing
            cudaCores = deviceProp.multiProcessorCount * 64;
        } else if (deviceProp.major == 8) { // Ampere
            cudaCores = deviceProp.multiProcessorCount * (deviceProp.minor == 9 ? 64 : 128); // A100 has 64 FP32/SM, GA10x gaming GPUs have 128 FP32 units/SM in a different config
        } else if (deviceProp.major == 9) { // Hopper
             cudaCores = deviceProp.multiProcessorCount * 128; // H100 has 128 FP32 CUDA cores per SM
        } else {
            cudaCores = deviceProp.multiProcessorCount; // Fallback or unknown
        }
        if (cudaCores > 0) {
            printf("  Approximate CUDA Cores:    %d\n", cudaCores);
        }

        printf("  Concurrent Kernels:        %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
        printf("  ECC Enabled:               %s\n", deviceProp.ECCEnabled ? "Yes" : "No");
        printf("------------------------------------------------------\n");
    }

    // Reset the device to free resources associated with this context (optional for simple query)
    // CUDA_CHECK(hipDeviceReset());

    return 0;
}
