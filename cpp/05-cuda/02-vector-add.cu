// This program implements vector addition on the GPU using CUDA.
// It demonstrates basic CUDA concepts: kernel definition, memory management,
// data transfers between host and device, and kernel launch.
//
// ================================= IMPORTANT ==================================
// TO COMPILE AND RUN THIS FILE:
// 1. CUDA Toolkit: You must have the NVIDIA CUDA Toolkit installed.
// 2. NVIDIA GPU: You need an NVIDIA CUDA-enabled GPU.
// 3. NVIDIA Driver: Ensure you have a compatible NVIDIA driver installed.
// 4. Compiler: Use the NVIDIA CUDA Compiler (nvcc).
//
// COMPILE COMMAND (example):
//   nvcc 02-vector-add.cu -o 02-vector-add
//
// RUN COMMAND (example):
//   ./02-vector-add
// ==============================================================================

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath> // For fabs in verification
#include <iomanip> // For std::fixed, std::setprecision

// --- CUDA Error Checking Macro ---
#define CUDA_CHECK(err) __cuda_check_errors((err), __FILE__, __LINE__)

inline void __cuda_check_errors(hipError_t err, const char *file, int line) {
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA error: %s (%d) in %s at line %d\n",
                hipGetErrorString(err), err, file, line);
        // For this example, exit on any CUDA error after the initial setup.
        exit(EXIT_FAILURE);
    }
}

// --- Vector Addition Kernel ---
// This function runs on the GPU (device code).
// `__global__` indicates it's a kernel callable from the host.
// It performs element-wise addition: c[i] = a[i] + b[i].
__global__ void vectorAddKernel(const float *a, const float *b, float *c, int n) {
    // Calculate the global thread ID for this thread.
    // - blockIdx.x: The ID of the current block in the grid (0-indexed).
    // - blockDim.x: The number of threads in each block (specified at launch).
    // - threadIdx.x: The ID of the current thread within its block (0-indexed).
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    // Check array bounds to ensure we don't write past the end of the arrays.
    // This is important if n is not perfectly divisible by threadsPerBlock.
    if (threadID < n) {
        c[threadID] = a[threadID] + b[threadID];
    }
}


int main() {
    std::cout << "--- CUDA Vector Addition ---" << std::endl;

    // --- 1. Initialization ---
    const int n = 1024 * 1024; // Number of elements in the vectors (e.g., 2^20)
    const size_t bytes = n * sizeof(float);
    std::cout << "Vector size: " << n << " elements (" << bytes / (1024.0 * 1024.0) << " MiB per vector)" << std::endl;

    // Allocate host memory (RAM) for vectors using std::vector for automatic memory management.
    std::vector<float> h_a(n);
    std::vector<float> h_b(n);
    std::vector<float> h_c(n); // Host vector to store the result from the device

    // Initialize host vectors h_a and h_b
    for (int i = 0; i < n; ++i) {
        h_a[i] = static_cast<float>(i) * 0.5f + 1.0f; // Example values
        h_b[i] = static_cast<float>(n - i) * 0.25f + 0.5f; // Example values
    }
    std::cout << "Host vectors initialized." << std::endl;

    // --- 2. Device Memory Allocation ---
    // Pointers for device memory (GPU VRAM)
    float *d_a = nullptr;
    float *d_b = nullptr;
    float *d_c = nullptr;

    // Allocate memory on the GPU device
    CUDA_CHECK(hipMalloc((void **)&d_a, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_b, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_c, bytes));
    std::cout << "Device memory allocated." << std::endl;

    // --- 3. Data Transfer (Host to Device) ---
    // Copy data from host vectors (h_a, h_b) to device vectors (d_a, d_b).
    // cudaMemcpyHostToDevice: Transfer data from RAM to GPU VRAM.
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice));
    std::cout << "Data transferred from host to device." << std::endl;

    // --- 4. Kernel Launch Configuration ---
    // Define the number of threads per block.
    // This is often chosen based on GPU architecture (e.g., multiples of warp size, typically 32).
    // Common values are 128, 256, 512, 1024. Max is usually 1024.
    const int threadsPerBlock = 256;

    // Calculate the number of blocks in the grid.
    // This ensures enough threads are launched to cover all 'n' elements.
    // The formula (n + threadsPerBlock - 1) / threadsPerBlock is a common idiom for ceiling division.
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "Kernel launch configuration: " << std::endl;
    std::cout << "  Threads per block: " << threadsPerBlock << std::endl;
    std::cout << "  Blocks per grid:   " << blocksPerGrid << std::endl;
    std::cout << "  Total threads:     " << threadsPerBlock * blocksPerGrid << " (may be > n)" << std::endl;


    // --- 5. Kernel Launch ---
    // Launch the vectorAddKernel on the GPU.
    // Syntax: kernel_name<<<blocksPerGrid, threadsPerBlock>>>(argument_list);
    std::cout << "Launching vectorAddKernel..." << std::endl;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Check for errors during kernel launch (asynchronous, so errors might appear later)
    CUDA_CHECK(hipGetLastError());
    // For robust synchronization and error checking after kernel, use cudaDeviceSynchronize.
    // This waits for all preceding device operations (including kernel) to complete.
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "Kernel execution finished." << std::endl;


    // --- 6. Data Transfer (Device to Host) ---
    // Copy the result vector (d_c) from device memory back to host memory (h_c).
    // cudaMemcpyDeviceToHost: Transfer data from GPU VRAM to RAM.
    CUDA_CHECK(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost));
    std::cout << "Result transferred from device to host." << std::endl;

    // --- 7. Verification (Optional but Recommended) ---
    std::cout << "Verifying result on the host..." << std::endl;
    bool success = true;
    float epsilon = 1e-5f; // Tolerance for floating point comparisons
    int errors_found = 0;
    int max_errors_to_print = 10;

    for (int i = 0; i < n; ++i) {
        float expected = h_a[i] + h_b[i];
        if (std::fabs(h_c[i] - expected) > epsilon) {
            success = false;
            if (errors_found < max_errors_to_print) {
                std::cerr << std::fixed << std::setprecision(6)
                          << "Verification FAILED at index " << i << ": "
                          << "Host_A=" << h_a[i] << ", Host_B=" << h_b[i]
                          << ", Expected=" << expected << ", GPU_Result=" << h_c[i]
                          << ", Diff=" << std::fabs(h_c[i] - expected) << std::endl;
            }
            errors_found++;
        }
    }

    if (success) {
        std::cout << "Verification PASSED! All elements match." << std::endl;
    } else {
        std::cout << "Verification FAILED. " << errors_found << " mismatch(es) found." << std::endl;
        if (errors_found > max_errors_to_print) {
            std::cout << "(Only first " << max_errors_to_print << " mismatches shown)" << std::endl;
        }
    }

    // --- 8. Cleanup ---
    // Free GPU device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    std::cout << "Device memory freed." << std::endl;

    // Host memory (h_a, h_b, h_c) is managed by std::vector and will be freed automatically.

    std::cout << "\nCUDA vector addition demonstration complete." << std::endl;

    // Optional: Reset CUDA device to clean up context resources, not strictly necessary here.
    // CUDA_CHECK(cudaDeviceReset());

    return 0;
}
